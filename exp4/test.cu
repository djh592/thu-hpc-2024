
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel()
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
    mykernel<<<2, 3>>>();
    hipDeviceSynchronize();
    return 0;
}